
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN

#include "./cudnn_softmax_layer.hpp"

namespace caffe {

void CuDNNSoftmaxLayer::Forward_gpu(const vector<Blob*>& bottom,
                                    const vector<Blob*>& top) {
  const real_t* bottom_data = bottom[0]->gpu_data();
  real_t* top_data = top[0]->mutable_gpu_data();
  CUDNN_CHECK(cudnnSoftmaxForward(handle_, CUDNN_SOFTMAX_ACCURATE,
              CUDNN_SOFTMAX_MODE_CHANNEL,
              cudnn::dataType<real_t>::one,
              bottom_desc_, bottom_data,
              cudnn::dataType<real_t>::zero,
              top_desc_, top_data));
}

}  // namespace caffe

#endif  // USE_CUDNN
